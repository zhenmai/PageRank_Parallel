#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream> // std::ifstream
#include <sstream> // std::stringstream
#include <string> // std::string, std::stoi
#include <cstring> // std::strcmp
#include <cmath>
#include <vector>
#include <chrono>
#include <ctime>
#include "Graph.hpp"

#define GPU 1

using namespace CSC586C::gpu_graph;

extern const double damping_factor = 0.85;
extern const unsigned max_iterations = 100;
extern const double tolerance = 1e-10;
const int blocksize = 512;

// Read Input (pairs of source and destination links) from file with format:
// src_index dest_index
// ... 
// src_index dest_index 
ColdEdge ReadInputFromTextFile(const char* input_file, unsigned& num_vertices)
{
    std::ifstream myfile (input_file);
    ColdEdge edges;
    unsigned source, destination;
    if (myfile.is_open()) 
    {
      while(myfile >> source >> destination)
      {
        unsigned larger = (source > destination)? source : destination;
        num_vertices = (num_vertices > larger)? num_vertices : larger;
        edges.src.push_back(source);
        edges.dest.push_back(destination);  
      }
      ++num_vertices;
      myfile.close();
    }
    return edges;
}

bool ToleranceCheck(const unsigned& num_v, HotData& hotData)
{
    // Sum up the pagerank
    double pr_sum = 0.0;
    for (unsigned i = 0; i < num_v; i++) 
    {
        pr_sum += hotData.pagerank[i];
    }
    // Calculate the cur_toleranceor
    pr_sum = 1.0 / pr_sum;
    double cur_tolerance = 0.0;
    for (unsigned i = 0; i < num_v; i++)
    {
        hotData.pagerank[i] *= pr_sum;
        // norm 1
        cur_tolerance += std::fabs(hotData.pagerank[i] - hotData.pre_pagerank[i]);
    }
    if (cur_tolerance < tolerance)
    {
        std::cout << "Current toleranceor: " << cur_tolerance << std::endl;
        return true;
    }
    return false;
}

#ifdef GPU
__global__
void update_pagerank( int *ingoing_edges_num, int *outgoing_edges_num,
                      int *begin_index, int *adj_edges, double *pre_pagerank,
                      double* pr_dangling, double* pr_random, double *pagerank,
                      size_t n )
{
   int const index = threadIdx.x + blockIdx.x * blockDim.x;
   if( index < n ) {
      int num_edges = ingoing_edges_num[index];
      int begin_index_ = begin_index[index];
      for( int i = 0; i < num_edges; ++i ){
         int inward_edge_index = adj_edges[begin_index_ + i];
         double pr_eigenvector = 0.85 * pre_pagerank[inward_edge_index]
                                 / outgoing_edges_num[inward_edge_index];
         pagerank[index] += pr_eigenvector;
      }
      pagerank[index] += (*pr_random + *pr_dangling);
   }
}
#endif

void PageRank(GPU_Graph *graph)
{
#ifdef GPU
   const unsigned num_v = graph->VertexesNum();
   double init_rank = double(1.0 / num_v);
   double pr_random = (1.0 - damping_factor) / num_v;

   // calculate number of blocks. block_size is fixed to 512
   auto const num_blocks = std::ceil( num_v / static_cast< float >( blocksize) );
   //Initialize all memories used by GPU
   int *dev_ingoing_edge_nums;
   int *dev_outgoing_edge_nums;
   int *dev_begin_index;
   int *dev_adj_edges;
   double *dev_pre_pagerank;
   double *dev_pagerank;
   double *dev_pr_dangling;
   double *dev_pr_random;

   //Allocate memory for elments mapped to GPU
   hipMalloc( (void **) &dev_ingoing_edge_nums, num_v*sizeof(int) );
   hipMalloc( (void **) &dev_outgoing_edge_nums, num_v*sizeof(int) );
   hipMalloc( (void **) &dev_begin_index, num_v*sizeof(int) );
   hipMalloc( (void **) &dev_adj_edges, graph->num_edges*sizeof(int) );
   hipMalloc( (void **) &dev_pagerank, num_v*sizeof(double) );
   hipMalloc( (void **) &dev_pre_pagerank, num_v*sizeof(double) );
   hipMalloc( (void **) &dev_pr_dangling, sizeof(double) );
   hipMalloc( (void **) &dev_pr_random, sizeof(double) );

   //Initialize obejcts that won't be changed in the algorithm
   hipMemcpy( dev_ingoing_edge_nums, graph->ingoing_edges_num.data(),
               num_v*sizeof(int), hipMemcpyHostToDevice );
   hipMemcpy( dev_outgoing_edge_nums, graph->hotData.outgoing_edges_num.data(),
               num_v*sizeof(int), hipMemcpyHostToDevice );
   hipMemcpy( dev_begin_index, graph->beginIndex.data(),
               num_v*sizeof(int), hipMemcpyHostToDevice );
   hipMemcpy( dev_adj_edges, graph->adjE,
               graph->num_edges*sizeof(int), hipMemcpyHostToDevice );
   hipMemcpy( dev_pr_random, &pr_random, sizeof(double), hipMemcpyHostToDevice );

   //Initialize objects that will be updated in the algorithm, and copy
   //them from host to device
   graph->hotData.pagerank.assign(num_v, init_rank);
   graph->hotData.pre_pagerank.assign(num_v, 0);
   hipMemcpy( dev_pre_pagerank, graph->hotData.pre_pagerank.data(),
               num_v*sizeof(double), hipMemcpyHostToDevice );
   hipMemcpy( dev_pagerank, graph->hotData.pagerank.data(),
               num_v*sizeof(double), hipMemcpyHostToDevice );

   unsigned iter = 0;
   while(iter++ < max_iterations){
      double dangling_pr_sum = 0.0;
      // Update the pagerank values in every iteration
      for (unsigned i = 0; i < num_v; i++)
      {
         dangling_pr_sum += graph->hotData.pagerank[i] * (graph->hotData.outgoing_edges_num[i] == 0);
         graph->hotData.pre_pagerank[i] = 0.0;
      }
      double pr_dangling = damping_factor * dangling_pr_sum / num_v;
      hipMemcpy( dev_pre_pagerank, graph->hotData.pagerank.data(),
                  num_v*sizeof(double), hipMemcpyHostToDevice );
      hipMemcpy( dev_pagerank, graph->hotData.pre_pagerank.data(),
                  num_v*sizeof(double), hipMemcpyHostToDevice );
      hipMemcpy( dev_pr_dangling, &pr_dangling, sizeof(double), hipMemcpyHostToDevice );

      //Main function in this algorithm to update pagerank at each iteration, hand over to GPU
      update_pagerank<<< num_blocks, blocksize >>>(dev_ingoing_edge_nums, dev_outgoing_edge_nums,
                                                   dev_begin_index, dev_adj_edges, dev_pre_pagerank,
                                                   dev_pr_dangling, dev_pr_random, dev_pagerank, num_v);
      hipMemcpy( graph->hotData.pagerank.data(), dev_pagerank, num_v*sizeof(double), hipMemcpyDeviceToHost );
      hipMemcpy( graph->hotData.pre_pagerank.data(), dev_pre_pagerank, num_v*sizeof(double), hipMemcpyDeviceToHost );
      // finish when cur_toleranceor is smaller than tolerance we set
      if(ToleranceCheck(num_v, graph->hotData)) 
      {
          std::cout << "Iteration time: " << iter << std::endl;
          break;
      }
   }
   // Free the memory on device side
   hipFree( dev_ingoing_edge_nums );
   hipFree( dev_outgoing_edge_nums );
   hipFree( dev_begin_index );
   hipFree( dev_adj_edges );
   hipFree( dev_pagerank );
   hipFree( dev_pre_pagerank );
   hipFree( dev_pr_dangling );
   hipFree( dev_pr_random );
#endif

#if 0
    // This is the original algorithm in CPU which we port to GPU
    const unsigned num_v = graph->VertexesNum();
    double init_rank = double(1.0 / num_v);
    double pr_random = (1.0 - damping_factor) / num_v;

    for (unsigned i = 0; i < num_v; i++)
    {
        graph->hotData.pagerank[i] = init_rank;
        graph->hotData.pre_pagerank[i] = 0.0;
    }

    unsigned iter = 0;
    while (iter++ < max_iterations)
    {
        double dangling_pr_sum = 0.0;
        // Update the pagerank values in every iteration
        for (unsigned i = 0; i < num_v; i++)
        {
            graph->hotData.pre_pagerank[i] = graph->hotData.pagerank[i];
            graph->hotData.pagerank[i] = 0.0;
            dangling_pr_sum += graph->hotData.pre_pagerank[i] * (graph->hotData.outgoing_edges_num[i] == 0);
        }

        double pr_dangling = damping_factor * dangling_pr_sum / num_v;

        // Iterater all the vertexes and calculate its adjacency function l(pi,pj) of all inward links
        // Update its pagerank value by adding pr_eigenvector from its inward links separately
        for( int i = 0; i < num_v; ++i )
        {
            unsigned inward_edges_num = graph->ingoing_edges_num[i];
            int begin_index = graph->beginIndex[i];
            for( int j = 0; j < inward_edges_num; ++j){
               unsigned inward_edge_index = graph->adjE[begin_index + j];
               double pr_eigenvector = damping_factor * graph->hotData.pre_pagerank[inward_edge_index]
                                        / graph->hotData.outgoing_edges_num[inward_edge_index];
               graph->hotData.pagerank[i] += pr_eigenvector;
            }
            graph->hotData.pagerank[i] += (pr_random + pr_dangling);
        }
        // finish when cur_toleranceor is smaller than tolerance we set
        if(ToleranceCheck(num_v, graph->hotData)) 
        {
            std::cout << "Iteration time: " << iter << std::endl;
            break;
        }
    }
#endif
}

void printFinalResults(GPU_Graph* graph)
{
    std::cout << "PageRank values: \n";
    for(int i = 0; i < graph->VertexesNum(); ++i)
    {
        std::cout << "The index is: " << i << " with value " << graph->hotData.pagerank[i] << '\n';  
    }
    std::cout<<'\n';
}

void PrintBenchmark(std::chrono::time_point<std::chrono::steady_clock> start_t, std::chrono::time_point<std::chrono::steady_clock> const end_t, const unsigned loop_t)
{
    auto const avg_time = std::chrono::duration_cast<std::chrono::microseconds>( end_t - start_t ).count() / double(loop_t);
    std::cout << "Average total running time  = " << avg_time << " us" << std::endl;
}

int main(int argc, char *argv[])
{
    unsigned loop_times = 10;
    unsigned num_vertices = 0;
    if(argc >= 4)
    {
        const char* test_mode = argv[2];

        ColdEdge input = ReadInputFromTextFile(argv[1], num_vertices);

        if(std::strcmp(test_mode, "total") == 0)
        {
            auto const start_time = std::chrono::steady_clock::now();

            for (int i = 0; i < loop_times; i++)
            {
                GPU_Graph graph(num_vertices, input);
                PageRank(&graph);
                //printFinalResults(&graph);
            }  
            auto const end_time = std::chrono::steady_clock::now(); 
            PrintBenchmark(start_time, end_time, loop_times);
        }
        else if(std::strcmp(test_mode, "graph") == 0 )
        {
            auto const start_time = std::chrono::steady_clock::now();
            GPU_Graph graph(num_vertices, input);
            auto const end_time = std::chrono::steady_clock::now(); 

            PageRank(&graph);  
            PrintBenchmark(start_time, end_time, 1);          
        }
        else if(std::strcmp(test_mode, "pagerank") == 0)
        {
            GPU_Graph graph(num_vertices, input);
            auto const start_time = std::chrono::steady_clock::now();
            for (unsigned i = 0; i < loop_times; i++)
            {
                PageRank(&graph);
            }
            auto const end_time = std::chrono::steady_clock::now(); 
            PrintBenchmark(start_time, end_time, loop_times);
        }
        else
        {
            std::cout << "Invalid Input!" << std::endl;
            std::cout << "Please input the input text file name wanted in argc[1]" << std::endl;
            std::cout << "Please input the time mode(total/graph/pangerank) to be record in argc[2]" << std::endl;
            std::cout << "Please input the number of threads wanted to use in argc[3]" << std::endl;
        }
    }
    else if (argc >= 2 && argc < 4)
    {
        ColdEdge input = ReadInputFromTextFile(argv[1], num_vertices);
        auto const start_time = std::chrono::steady_clock::now();
        for (int i = 0; i < loop_times; i++)
        {
            GPU_Graph graph(num_vertices, input);
            PageRank(&graph);
        }  
        auto const end_time = std::chrono::steady_clock::now(); 
        PrintBenchmark(start_time, end_time, loop_times);
    }
    else
    {
        std::cout << "Invalid Input: " << std::endl;
        std::cout << "Please input the input text file name wanted in argc[1]" << std::endl;
        std::cout << "Please input the time mode(total/graph/pangerank) to be record in argc[2]" << std::endl;
        std::cout << "Please input the number of threads wanted to use in argc[3]" << std::endl;
    }
    return 0;
}
